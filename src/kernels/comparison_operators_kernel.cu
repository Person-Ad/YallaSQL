#include "hip/hip_runtime.h"
#include "kernels/comparison_operators_kernel.hpp"

namespace YallaSQL::Kernel {
    //! 
    template <typename T, typename Op>
    __global__ void outer_join_batches(T* __restrict__ rhs, T* __restrict__ lhs, 
                                        uint32_t* pairs, 
                                        int* actualSz,
                                        unsigned int left_rows,
                                        unsigned int right_rows) {
        unsigned int lidx = threadIdx.x + blockIdx.x * blockDim.x;
        unsigned int ridx = threadIdx.y + blockIdx.y * blockDim.y;
        Op op;
        if (lidx < left_rows && ridx < right_rows) {
            bool match = op.apply(lhs[lidx], rhs[ridx]);
            if (match) {
                unsigned int write_pos = atomicAdd(actualSz, 1);
                pairs[write_pos * 2] = lidx;
                pairs[write_pos * 2 + 1] = ridx;
            }
        }
    }

    __global__ void and_join_batches(const uint32_t* __restrict__ l_pairs, const uint32_t* __restrict__ r_pairs, 
                                        uint32_t* pairs, // out pairs
                                        uint32_t* mask, // mark if I should or already written once // bitset
                                        int* actualSz,
                                        unsigned int left_rows,
                                        unsigned int right_rows) {
        unsigned int lidx = threadIdx.x + blockIdx.x * blockDim.x;
        unsigned int ridx = threadIdx.y + blockIdx.y * blockDim.y;

        if (lidx < left_rows && ridx < right_rows) {
            // Check if pair in l_pairs matches pair in r_pairs
            bool match = (l_pairs[2 * lidx] == r_pairs[2 * ridx]) && 
                         (l_pairs[2 * lidx + 1] == r_pairs[2 * ridx + 1]);
    
            if (match) {
                // Use atomic operation to set bit in mask to avoid duplicates
                unsigned int word_idx = lidx / 32; // Each unsigned int holds 32 bits
                unsigned int bit_idx = lidx % 32;
                unsigned int bit_mask = 1U << bit_idx;
                unsigned int prev = atomicOr(&mask[word_idx], bit_mask);
    
                // Check if bit was already set (pair was written before)
                if (!(prev & bit_mask)) {
                    // Pair is new, write to output
                    unsigned int write_pos = atomicAdd(actualSz, 1);
                    pairs[2 * write_pos] = l_pairs[2 * lidx];
                    pairs[2 * write_pos + 1] = l_pairs[2 * lidx + 1];
                }
            }
        }
    }


    template <typename T, typename Op>
    __global__ void apply_batches(T* __restrict__ rhs, T* __restrict__ lhs, bool* res, const unsigned int sz, bool isneg) {
        unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
        unsigned int stride = blockDim.x * gridDim.x;
    
        Op op;

        #pragma unroll 
        for (int k = 0; k < COARSENING_FACTOR; k++) {
            unsigned int global_idx = idx + k * stride;
            if (global_idx < sz) {
                res[global_idx] = isneg ? !op.apply(lhs[global_idx], rhs[global_idx]) : op.apply(lhs[global_idx], rhs[global_idx]);
            }
        }

    }


    template <typename T, typename Op>
    __global__ void apply_batches_scalar_rhs(T* __restrict__ rhs, T* __restrict__ lhs, bool* res, const unsigned int sz, bool isneg) {
        unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
        unsigned int stride = blockDim.x * gridDim.x;
    
        Op op;

        #pragma unroll 
        for (int k = 0; k < COARSENING_FACTOR; k++) {
            unsigned int global_idx = idx + k * stride;
            if (global_idx < sz) {
                res[global_idx] = isneg ? !op.apply(lhs[global_idx], rhs[0]) : op.apply(lhs[global_idx], rhs[0]);
            }
        }

    }

    template <typename T, typename Op>
    __global__ void apply_batches_scalar_lhs(T* __restrict__ rhs, T* __restrict__ lhs, bool* res, const unsigned int sz, bool isneg) {
        unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
        unsigned int stride = blockDim.x * gridDim.x;
    
        Op op;

        #pragma unroll 
        for (int k = 0; k < COARSENING_FACTOR; k++) {
            unsigned int global_idx = idx + k * stride;
            if (global_idx < sz) {
                res[global_idx] = isneg ? !op.apply(lhs[0], rhs[global_idx]) : op.apply(lhs[0], rhs[global_idx]);
            }
        }

    }
    


    template <typename T, typename Op>
    void launch_conditional_operators(T* __restrict__ d_rhs, T* __restrict__ d_lhs, OperandType t_rhs, OperandType t_lhs, bool* d_res, unsigned int sz, hipStream_t& stream, bool isneg) {
        if(t_rhs == OperandType::SCALAR && t_lhs == OperandType::SCALAR) {
            apply_batches<T, Op><<<1, 1>>>(d_rhs, d_lhs, d_res, sz, isneg);
            return;
        } 
        
        dim3 threads(BLOCK_DIM);
        dim3 blocks (CEIL_DIV(sz, threads.x * COARSENING_FACTOR));
        
        if(t_rhs == OperandType::SCALAR) 
            apply_batches_scalar_rhs<T, Op><<<blocks, threads, 0, stream>>>(d_rhs, d_lhs, d_res, sz, isneg);
        else if(t_lhs == OperandType::SCALAR)
            apply_batches_scalar_lhs<T, Op><<<blocks, threads, 0, stream>>>(d_rhs, d_lhs, d_res, sz, isneg);
        else
            apply_batches<T, Op><<<blocks, threads, 0, stream>>>(d_rhs, d_lhs, d_res, sz, isneg);

        CUDA_CHECK_LAST();
    }


    template <typename T, typename Op>
    void launch_outer_join_operators(T* __restrict__ d_rhs, T* __restrict__ d_lhs, 
                                   uint32_t* pairs, 
                                   int* actualSz,
                                   unsigned int left_rows,
                                   unsigned int right_rows,
                                   hipStream_t stream) {
        // unsigned int total_size = left_rows * right_rows;
        
        dim3 threads(32, 32);
        dim3 blocks(CEIL_DIV(left_rows, threads.x), CEIL_DIV(right_rows, threads.y));
        
        outer_join_batches<T, Op><<<blocks, threads, 0, stream>>>(d_rhs, d_lhs, pairs, actualSz, left_rows, right_rows);

        CUDA_CHECK_LAST();
    }

    void launch_and_join_operators(const uint32_t* __restrict__ l_pairs, const uint32_t* __restrict__ r_pairs, 
                                    uint32_t* pairs, // out pairs
                                    uint32_t* mask, // mark if I should or already written once // bitset
                                    int* actualSz,
                                    unsigned int left_rows,
                                    unsigned int right_rows,
                                   hipStream_t stream) {
        // unsigned int total_size = left_rows * right_rows;
        
        dim3 threads(32, 32);
        dim3 blocks(CEIL_DIV(left_rows, threads.x), CEIL_DIV(right_rows, threads.y));
        
        and_join_batches<<<blocks, threads, 0, stream>>>(l_pairs, r_pairs, pairs, mask, actualSz, left_rows, right_rows);

        CUDA_CHECK_LAST();
    }
   // Explicit instantiations with __restrict__
    template void launch_conditional_operators<int, LEOperator<int>>(int* __restrict__, int* __restrict__, OperandType, OperandType, bool*, unsigned int, hipStream_t&, bool);
    template void launch_conditional_operators<int, GEOperator<int>>(int* __restrict__, int* __restrict__, OperandType, OperandType, bool*, unsigned int, hipStream_t&, bool);
    template void launch_conditional_operators<int, LEQOperator<int>>(int* __restrict__, int* __restrict__, OperandType, OperandType, bool*, unsigned int, hipStream_t&, bool);
    template void launch_conditional_operators<int, GEQOperator<int>>(int* __restrict__, int* __restrict__, OperandType, OperandType, bool*, unsigned int, hipStream_t&, bool);
    template void launch_conditional_operators<int, EQOperator<int>>(int* __restrict__, int* __restrict__, OperandType, OperandType, bool*, unsigned int, hipStream_t&, bool);
    template void launch_conditional_operators<int, NEQOperator<int>>(int* __restrict__, int* __restrict__, OperandType, OperandType, bool*, unsigned int, hipStream_t&, bool);

    template void launch_conditional_operators<float, LEOperator<float>>(float* __restrict__, float* __restrict__, OperandType, OperandType, bool*, unsigned int, hipStream_t&, bool);
    template void launch_conditional_operators<float, GEOperator<float>>(float* __restrict__, float* __restrict__, OperandType, OperandType, bool*, unsigned int, hipStream_t&, bool);
    template void launch_conditional_operators<float, LEQOperator<float>>(float* __restrict__, float* __restrict__, OperandType, OperandType, bool*, unsigned int, hipStream_t&, bool);
    template void launch_conditional_operators<float, GEQOperator<float>>(float* __restrict__, float* __restrict__, OperandType, OperandType, bool*, unsigned int, hipStream_t&, bool);
    template void launch_conditional_operators<float, EQOperator<float>>(float* __restrict__, float* __restrict__, OperandType, OperandType, bool*, unsigned int, hipStream_t&, bool);
    template void launch_conditional_operators<float, NEQOperator<float>>(float* __restrict__, float* __restrict__, OperandType, OperandType, bool*, unsigned int, hipStream_t&, bool);

    template void launch_conditional_operators<int64_t, LEOperator<int64_t>>(int64_t* __restrict__, int64_t* __restrict__, OperandType, OperandType, bool*, unsigned int, hipStream_t&, bool);
    template void launch_conditional_operators<int64_t, GEOperator<int64_t>>(int64_t* __restrict__, int64_t* __restrict__, OperandType, OperandType, bool*, unsigned int, hipStream_t&, bool);
    template void launch_conditional_operators<int64_t, LEQOperator<int64_t>>(int64_t* __restrict__, int64_t* __restrict__, OperandType, OperandType, bool*, unsigned int, hipStream_t&, bool);
    template void launch_conditional_operators<int64_t, GEQOperator<int64_t>>(int64_t* __restrict__, int64_t* __restrict__, OperandType, OperandType, bool*, unsigned int, hipStream_t&, bool);
    template void launch_conditional_operators<int64_t, EQOperator<int64_t>>(int64_t* __restrict__, int64_t* __restrict__, OperandType, OperandType, bool*, unsigned int, hipStream_t&, bool);
    template void launch_conditional_operators<int64_t, NEQOperator<int64_t>>(int64_t* __restrict__, int64_t* __restrict__, OperandType, OperandType, bool*, unsigned int, hipStream_t&, bool);

    template void launch_conditional_operators<String, LEOperator<String>>(String* __restrict__, String* __restrict__, OperandType, OperandType, bool*, unsigned int, hipStream_t&, bool);
    template void launch_conditional_operators<String, GEOperator<String>>(String* __restrict__, String* __restrict__, OperandType, OperandType, bool*, unsigned int, hipStream_t&, bool);
    template void launch_conditional_operators<String, LEQOperator<String>>(String* __restrict__, String* __restrict__, OperandType, OperandType, bool*, unsigned int, hipStream_t&, bool);
    template void launch_conditional_operators<String, GEQOperator<String>>(String* __restrict__, String* __restrict__, OperandType, OperandType, bool*, unsigned int, hipStream_t&, bool);
    template void launch_conditional_operators<String, EQOperator<String>>(String* __restrict__, String* __restrict__, OperandType, OperandType, bool*, unsigned int, hipStream_t&, bool);
    template void launch_conditional_operators<String, NEQOperator<String>>(String* __restrict__, String* __restrict__, OperandType, OperandType, bool*, unsigned int, hipStream_t&, bool);


    template void launch_conditional_operators<bool, ANDOperator>(bool* __restrict__, bool* __restrict__, OperandType, OperandType, bool*, unsigned int, hipStream_t&, bool);
    template void launch_conditional_operators<bool, OROperator>(bool* __restrict__, bool* __restrict__, OperandType, OperandType, bool*, unsigned int, hipStream_t&, bool);


    
    template void launch_outer_join_operators<int,      EQOperator<int>>(int* __restrict__ , int* __restrict__, uint32_t*, int*, unsigned int,unsigned int, hipStream_t);
    template void launch_outer_join_operators<float,    EQOperator<float>>(float* __restrict__ , float* __restrict__, uint32_t*, int*, unsigned int,unsigned int, hipStream_t);
    template void launch_outer_join_operators<int64_t,  EQOperator<int64_t>>(int64_t* __restrict__ , int64_t* __restrict__, uint32_t*, int*, unsigned int,unsigned int, hipStream_t);
    template void launch_outer_join_operators<String,   EQOperator<String>>(String* __restrict__ , String* __restrict__, uint32_t*, int*, unsigned int,unsigned int, hipStream_t);

    template void launch_outer_join_operators<int,      NEQOperator<int>>(int* __restrict__ , int* __restrict__, uint32_t*, int*, unsigned int,unsigned int, hipStream_t);
    template void launch_outer_join_operators<float,    NEQOperator<float>>(float* __restrict__ , float* __restrict__, uint32_t*, int*, unsigned int,unsigned int, hipStream_t);
    template void launch_outer_join_operators<int64_t,  NEQOperator<int64_t>>(int64_t* __restrict__ , int64_t* __restrict__, uint32_t*, int*, unsigned int,unsigned int, hipStream_t);
    template void launch_outer_join_operators<String,   NEQOperator<String>>(String* __restrict__ , String* __restrict__, uint32_t*, int*, unsigned int,unsigned int, hipStream_t);

    template void launch_outer_join_operators<int,      LEOperator<int>>(int* __restrict__ , int* __restrict__, uint32_t*, int*, unsigned int,unsigned int, hipStream_t);
    template void launch_outer_join_operators<float,    LEOperator<float>>(float* __restrict__ , float* __restrict__, uint32_t*, int*, unsigned int,unsigned int, hipStream_t);
    template void launch_outer_join_operators<int64_t,  LEOperator<int64_t>>(int64_t* __restrict__ , int64_t* __restrict__, uint32_t*, int*, unsigned int,unsigned int, hipStream_t);
    template void launch_outer_join_operators<String,   LEOperator<String>>(String* __restrict__ , String* __restrict__, uint32_t*, int*, unsigned int,unsigned int, hipStream_t);

    template void launch_outer_join_operators<int,      LEQOperator<int>>(int* __restrict__ , int* __restrict__, uint32_t*, int*, unsigned int,unsigned int, hipStream_t);
    template void launch_outer_join_operators<float,    LEQOperator<float>>(float* __restrict__ , float* __restrict__, uint32_t*, int*, unsigned int,unsigned int, hipStream_t);
    template void launch_outer_join_operators<int64_t,  LEQOperator<int64_t>>(int64_t* __restrict__ , int64_t* __restrict__, uint32_t*, int*, unsigned int,unsigned int, hipStream_t);
    template void launch_outer_join_operators<String,   LEQOperator<String>>(String* __restrict__ , String* __restrict__, uint32_t*, int*, unsigned int,unsigned int, hipStream_t);
    
    template void launch_outer_join_operators<int,      GEQOperator<int>>(int* __restrict__ , int* __restrict__, uint32_t*, int*, unsigned int,unsigned int, hipStream_t);
    template void launch_outer_join_operators<float,    GEQOperator<float>>(float* __restrict__ , float* __restrict__, uint32_t*, int*, unsigned int,unsigned int, hipStream_t);
    template void launch_outer_join_operators<int64_t,  GEQOperator<int64_t>>(int64_t* __restrict__ , int64_t* __restrict__, uint32_t*, int*, unsigned int,unsigned int, hipStream_t);
    template void launch_outer_join_operators<String,   GEQOperator<String>>(String* __restrict__ , String* __restrict__, uint32_t*, int*, unsigned int,unsigned int, hipStream_t);

    template void launch_outer_join_operators<int,      GEOperator<int>>(int* __restrict__ , int* __restrict__, uint32_t*, int*, unsigned int,unsigned int, hipStream_t);
    template void launch_outer_join_operators<float,    GEOperator<float>>(float* __restrict__ , float* __restrict__, uint32_t*, int*, unsigned int,unsigned int, hipStream_t);
    template void launch_outer_join_operators<int64_t,  GEOperator<int64_t>>(int64_t* __restrict__ , int64_t* __restrict__, uint32_t*, int*, unsigned int,unsigned int, hipStream_t);
    template void launch_outer_join_operators<String,   GEOperator<String>>(String* __restrict__ , String* __restrict__, uint32_t*, int*, unsigned int,unsigned int, hipStream_t);

    


}